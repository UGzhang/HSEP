#include "Atom.cuh"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

// Initialize managed CUDA memory
void initMemory(Atom* atom, int N, int nrCells){
    atom->n = N;
    CHECK(hipMallocManaged(&atom->m, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->x, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->y, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->z, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->vx, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->vy, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->vz, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->fx, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->fy, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->fz, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->r, N * sizeof(real)));
    CHECK(hipMallocManaged(&atom->cells, nrCells * nrCells * nrCells * sizeof(int)));
    CHECK(hipMallocManaged(&atom->particles, N * sizeof(int)));
    CHECK(hipMallocManaged(&atom->I, N * 9 * sizeof(real))); // Added for inertia tensors
    CHECK(hipMallocManaged(&atom->tx, N * sizeof(real))); // Added for torques
    CHECK(hipMallocManaged(&atom->ty, N * sizeof(real))); // Added for torques
    CHECK(hipMallocManaged(&atom->tz, N * sizeof(real))); // Added for torques
    CHECK(hipMallocManaged(&atom->wx, N * sizeof(real))); // Added for angular velocities
    CHECK(hipMallocManaged(&atom->wy, N * sizeof(real))); // Added for angular velocities
    CHECK(hipMallocManaged(&atom->wz, N * sizeof(real))); // Added for angular velocities
    CHECK(hipMallocManaged(&atom->qx, N * sizeof(real))); // Added for quaternions
    CHECK(hipMallocManaged(&atom->qy, N * sizeof(real))); // Added for quaternions
    CHECK(hipMallocManaged(&atom->qz, N * sizeof(real))); // Added for quaternions
    CHECK(hipMallocManaged(&atom->qw, N * sizeof(real))); // Added for quaternions
}

// Load data to GPU
void prefetchGPU(Atom* atom, int N, int nrCells, hipDevice_t device){
    CHECK(hipMemPrefetchAsync(atom, sizeof(Atom), device, 0));
    CHECK(hipMemPrefetchAsync(atom->m, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->x, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->y, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->z, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->vx, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->vy, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->vz, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->fx, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->fy, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->fz, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->r, N * sizeof(real), device, 0));
    CHECK(hipMemPrefetchAsync(atom->cells, nrCells*nrCells*nrCells * sizeof(int), device, 0));
    CHECK(hipMemPrefetchAsync(atom->particles, N * sizeof(int), device, 0));
    CHECK(hipMemPrefetchAsync(atom->I, N * 9 * sizeof(real), device)); // Added for inertia tensors
    CHECK(hipMemPrefetchAsync(atom->tx, N * sizeof(real), device, 0)); // Added for torques
    CHECK(hipMemPrefetchAsync(atom->ty, N * sizeof(real), device, 0)); // Added for torques
    CHECK(hipMemPrefetchAsync(atom->tz, N * sizeof(real), device, 0)); // Added for torques
    CHECK(hipMemPrefetchAsync(atom->wx, N * sizeof(real), device, 0)); // Added for angular velocities
    CHECK(hipMemPrefetchAsync(atom->wy, N * sizeof(real), device, 0)); // Added for angular velocities
    CHECK(hipMemPrefetchAsync(atom->wz, N * sizeof(real), device, 0)); // Added for angular velocities
    CHECK(hipMemPrefetchAsync(atom->qx, N * sizeof(real), device, 0)); // Added for quaternions
    CHECK(hipMemPrefetchAsync(atom->qy, N * sizeof(real), device, 0)); // Added for quaternions
    CHECK(hipMemPrefetchAsync(atom->qz, N * sizeof(real), device, 0)); // Added for quaternions
    CHECK(hipMemPrefetchAsync(atom->qw, N * sizeof(real), device, 0)); // Added for quaternions
}

// Load data to CPU
void prefetchCPU(Atom* atom, Atom* atomCPU, int N, int nrCells){
    CHECK(hipMemcpyAsync(atomCPU, atom, sizeof(Atom), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->m, atom->m, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->x, atom->x, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->y, atom->y, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->z, atom->z, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->vx, atom->vx, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->vy, atom->vy, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->vz, atom->vz, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->fx, atom->fx, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->fy, atom->fy, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->fz, atom->fz, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->r, atom->r, N * sizeof(real), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->cells, atom->cells, nrCells*nrCells*nrCells * sizeof(int), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->particles, atom->particles, N * sizeof(int), hipMemcpyDeviceToHost, 0));
    CHECK(hipMemcpyAsync(atomCPU->I, atom->I, N * 9 * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for inertia tensors
    CHECK(hipMemcpyAsync(atomCPU->tx, atom->tx, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for torques
    CHECK(hipMemcpyAsync(atomCPU->ty, atom->ty, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for torques
    CHECK(hipMemcpyAsync(atomCPU->tz, atom->tz, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for torques
    CHECK(hipMemcpyAsync(atomCPU->wx, atom->wx, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for angular velocities
    CHECK(hipMemcpyAsync(atomCPU->wy, atom->wy, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for angular velocities
    CHECK(hipMemcpyAsync(atomCPU->wz, atom->wz, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for angular velocities
    CHECK(hipMemcpyAsync(atomCPU->qx, atom->qx, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for quaternions
    CHECK(hipMemcpyAsync(atomCPU->qy, atom->qy, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for quaternions
    CHECK(hipMemcpyAsync(atomCPU->qz, atom->qz, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for quaternions
    CHECK(hipMemcpyAsync(atomCPU->qw, atom->qw, N * sizeof(real), hipMemcpyDeviceToHost, 0)); // Added for quaternions
}

// Free memory
void freeMemory(Atom* atom){
    hipFree(atom->m);
    hipFree(atom->x);
    hipFree(atom->y);
    hipFree(atom->z);
    hipFree(atom->vx);
    hipFree(atom->vy);
    hipFree(atom->vz);
    hipFree(atom->fx);
    hipFree(atom->fy);
    hipFree(atom->fz);
    hipFree(atom->r);
    hipFree(atom->cells);
    hipFree(atom->particles);
    hipFree(atom->I); // Added for inertia tensors
    hipFree(atom->tx); // Added for torques
    hipFree(atom->ty); // Added for torques
    hipFree(atom->tz); // Added for torques
    hipFree(atom->wx); // Added for angular velocities
    hipFree(atom->wy); // Added for angular velocities
    hipFree(atom->wz); // Added for angular velocities
    hipFree(atom->qx); // Added for quaternions
    hipFree(atom->qy); // Added for quaternions
    hipFree(atom->qz); // Added for quaternions
    hipFree(atom->qw); // Added for quaternions
}

// Setting up the domain and particles
void initAtom(const char* filePath, Atom* atom, int nrCells, real r){
    std::string line;
    std::ifstream filestr;
    filestr.open(filePath, std::ios::in);
    // Check if the file was opened successfully
    if (!filestr.is_open()) {
        std::cerr << "Failed to open the file: " << filePath << std::endl;
        return;
    }
    int numPoints = 0;
    while (std::getline(filestr, line))
    {
        if (line.find("POINTS") != std::string::npos) {
            std::istringstream iss(line);
            std::string keyword;

            iss >> keyword >> numPoints;
            initMemory(atom, numPoints, nrCells);
            atom->n = numPoints;
            for(int i = 0; i < nrCells * nrCells * nrCells; i++){
                atom->cells[i] = -1;
            }
            for (int i = 0; i < numPoints; ++i) {
                std::getline(filestr, line);
                std::istringstream pointIss(line);
                pointIss >> atom->x[i] >> atom->y[i] >> atom->z[i];
            }
        } else if(line.find("LOOKUP_TABLE") != std::string::npos ){
            for (int i = 0; i < numPoints; ++i) {
                std::getline(filestr, line);
                std::istringstream pointIss(line);
                pointIss >> atom->m[i];
            }
        }else if(line.find("VECTORS") != std::string::npos ){
            for (int i = 0; i < numPoints; ++i) {
                std::getline(filestr, line);
                std::istringstream pointIss(line);
                pointIss >> atom->vx[i] >> atom->vy[i] >> atom->vz[i];
            }
        }

    }
    filestr.close();

    //initial the rest variables
    for (int i = 0; i < numPoints; ++i) {
        atom->fx[i] = 0.0;
        atom->fy[i] = 0.0;
        atom->fz[i] = 0.0;
        atom->r[i] = r;

        atom->tx[i] = 0.0; // Initialize torques
        atom->ty[i] = 0.0; // Initialize torques
        atom->tz[i] = 0.0; // Initialize torques
        atom->wx[i] = 0.0; // Initialize angular velocities
        atom->wy[i] = 0.0; // Initialize angular velocities
        atom->wz[i] = 0.0; // Initialize angular velocities

        // Initialize the inertia tensor for each sphere
        real mass = atom->m[i];
        real radius = atom->r[i];
        real inertia = (2.0 / 5.0) * mass * radius * radius;
        atom->I[i * 9 + 0] = inertia; // Ixx
        atom->I[i * 9 + 4] = inertia; // Iyy
        atom->I[i * 9 + 8] = inertia; // Izz
        atom->I[i * 9 + 1] = atom->I[i * 9 + 2] = atom->I[i * 9 + 3] = 0.0;
        atom->I[i * 9 + 5] = atom->I[i * 9 + 6] = atom->I[i * 9 + 7] = 0.0;

        // Initialize quaternions to identity (0,0,0,1)
        atom->qx[i] = 0.0;
        atom->qy[i] = 0.0;
        atom->qz[i] = 0.0;
        atom->qw[i] = 1.0;
    }
}